#include "hip/hip_runtime.h"
//	author: Benedykt Bela

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <time.h>
#include <stdio.h>
#include <stdio.h>

using namespace std;



//		funkcja wykonujaca sie na GPU 
//		argumenty to zadana liczba, dzielnik od ktorego ma sie zaczynac wykonywanie
//		tego wywolania funkcji oraz zmienna czy przyjmujaca wartosc 1 jezeli znalezlismy
//		dzielnik naszej liczby
__global__  void divide(long long int *liczba, int *dzielnik, int *czy)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	//	w kolejnych watkach dzielnik zwiekszam o dwa
	int abc = dzielnik[0] + 2 * i;

	//	jezeli znalezlismy dzielnik, to zmieniamy wartosc zmiennej czy na 1
	if (liczba[0] % abc == 0)
	{
		czy[0] = 1;
	}
}



//	funkca sprawdzajaca czy zadana liczba jest liczba pierwsza bez 
//	korzystania z GPU
int czy_pierwszaCPU(long long int liczba)
{
	//	licze pierwiastek z danej liczby, bo przeszukiwanie dzielnikow wiekszych
	//	jest bezcelowe
	long int pomocnicza = (int)sqrt(liczba);


	//	na poczatku sprawdzam czy liczba to 2 - wtedy zwracam, ze jest to liczba pierwsza
	if (liczba == 2)
	{
		cout << "liczba pierwsza" << endl;
	}
	//	jezeli nie jest to liczba 2, to sprawdzam czy jest parzysta, bo wtedy 
	//	wiadomo, ze jest zlozona
	else if (liczba % 2 == 0)
	{
		cout << "liczba zlozona" << endl;
	}
	//	jezeli nie jest ani dwojka, ani liczba parzysta
	else
	{
		//	dla kazdej nieparzystej liczby wiekszej od 2 i mniejszej od 
		// 	pierwiastka kwadratowego z liczby sprawdzanej
		for (int i = 3; i <= pomocnicza; i += 2)
		{
			//	jezeli i dzieli liczbe przeszukiwana bez reszty, jest to liczba zlozona
			if (liczba % i == 0)
			{
				cout << "liczba zlozona" << endl;

				return 0;
			}
		}

		//	jezeli powyzszy for sprawdzil wszystkie potencjalne dzielniki i nie znalazl
		//	dzielnika, to przeszukiwana liczba jest liczba pierwsza
		cout << "liczba pierwsza" << endl;
	}


	return 0;
}



//	funkcja sprawdzajaca czy zadana liczba jest liczba  pierwsza   
//	korzystajac z GPU
//		nieopisane fragmenty jak w funkcji czy_pierwszaCPU()
int czy_pierwszaGPU(long long int liczba_wczytana)
{
	//	generuje  wskazniki na liczby  przeszukiwane  i zmienna  czy dla CPU i GPU oraz
	//	size - rozmiar long long int'a
	long long int *liczba = new long long int;
	long long int *d_liczba = new long long int;
	int size = sizeof(long long int);
	int *czy = new int;
	int *d_czy = new int;

	//	nasze zaalokowane miejsce przyjmuje wartosc liczby wczytanej
	liczba[0] = liczba_wczytana;

	//	na GPU alokujemy pamiec dla zmiennej czy oraz liczby przeszukiwanej oraz kopiujemy
	//	wartosci tych zmiennych z CPU
	hipMalloc(&d_czy, sizeof(int));
	hipMalloc(&d_liczba, size);
	hipMemcpy(d_liczba, liczba, size, hipMemcpyHostToDevice);
	hipMemcpy(d_czy, czy, sizeof(int), hipMemcpyHostToDevice);

	long int pomocnicza = (int)sqrt(*liczba);


	if (liczba[0] == 2)
	{
		cout << "liczba pierwsza" << endl;
	}
	else if (liczba[0] % 2 == 0)
	{
		cout << "liczba zlozona" << endl;
	}
	else
	{
		//	alokacja zmiennej i do iterowania w ponizszej petli for na CPU i GPU
		int *i = new int;
		int *d_i = new int;
		hipMalloc(&d_i, sizeof(int));
	

		//	dla kazdego nieparzystego i mniejszego od pierwiastka kwadratowego z liczby przeszukiwanej
		for (i[0] = 3; i[0] <= pomocnicza;)
		{
			//	kopiuje wartosc zmiennej i na GPU
			hipMemcpy(d_i, i, sizeof(int), hipMemcpyHostToDevice);


			//	w zaleznosci od tego jak duza jest przeszukiwana liczba, a konkretnie jej pierwiastek -
			//	wywoluje funkcje divide wykonywana na GPU z roznymi parametrami
			if (pomocnicza > 100000)
			{
				divide << <2048, 128 >> > (d_liczba, d_i, d_czy);
				//	zwiekszamy poczatkowy indeks przeszukiwan na ostatni przeszukany na GPU
				i[0] += (2048 * 128 * 2);
			}	
			else if (pomocnicza > 10000)
			{
				divide << <256, 32 >> > (d_liczba, d_i, d_czy);
				i[0] += (256 * 32 * 2);
			}
			else if (pomocnicza > 200)
			{
				divide << <2, 32 >> > (d_liczba, d_i, d_czy);
				i[0] += (2 * 32 * 2);
			}
			else
			{
				divide << <1, 1 >> > (d_liczba, d_i, d_czy);
				i[0] += 2;
			}

			//	kopiujemy z GPU wartosc zmiennej czy
			hipMemcpy(czy, d_czy, sizeof(int), hipMemcpyDeviceToHost);

			//	jezeli GPU zmienila wartosc zmiennej czy na 1 oznacza to, ze znalazla dzielnik 
			//	liczby przeszukiwanej, wiec liczba jest zlozona i konczymy funkcje
			if (czy[0] == 1)
			{
				cout << "liczba zlozona" << endl;

				return 0;
			}
		}


		cout << "liczba pierwsza" << endl;
	}


	return 0;
}



int main()
{
	//	znak oraz czy jest pomocny przy obsludze funkcji, a liczba to liczba przeszukiwana
	char znak = 'X';
	bool czy = true;
	long long int liczba;


	//	petla wykonuje sie dopoki zmienna czy ma wartosc true, czyli do momentu wpisania 
	//	litery K od koniec jako zmienna wejsciowa
	while (czy)
	{
		//	podajemy przeszukiwana  liczbe
		cout << "Podaj liczbe... " << endl;
		cin >> liczba;

		//	inicjuje zmienne potrzebne do obliczania czasu wykonywania funkcji na GPU i CPU
		double start, end;


		//	mierze czas w chwili startu, wykonuje funkce na CPU, sprawdzam czas w momencie 
		//	zakonczenia funkcji i wyswietlam roznice tych wartosci, czyli czas wykonania funkcji
		start = clock();
		czy_pierwszaCPU(liczba);
		end = clock();
		cout << "Czas wykonania na CPU: " <<end - start << endl;


		//	jak wyzej, tylko dla GPU
		start = clock();
		czy_pierwszaGPU(liczba);
		end = clock();
		cout << "Czas wykonania na GPU: " << end - start << endl << endl;


		//	ponizsza zmienna znak i petla while sluza do obslugi programu:
		//		- jezeli wpiszemy znak 'C' program pyta nas o kolejna liczbe do sprawdzenia
		//		- jezeli wpiszemy znak 'K' program konczy dzialanie
		//		- jezeli wpisany znak jest inny niz K lub C program pyta nas ponownie
		znak = 'X';

		while (znak != 'K' && znak != 'C')
		{
			cout << "to end press K\nto continue press C" << endl;
			cin >> znak;

			if (znak == 'K')
				czy = false;
		}
	}


	return 0;
}
